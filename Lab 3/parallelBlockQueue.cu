#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "gputimer.h"

int read_input_one_two_four(int **input1, char* filepath){
 FILE* fp = fopen(filepath, "r");
    if (fp == NULL){
     fprintf(stderr, "Couldn't open file for reading\n");
     exit(1);
    }

    int counter = 0;
    int len;
    int length = fscanf(fp, "%d", &len);
    *input1 = ( int *)malloc(len * sizeof(int));

    int temp1;

    while (fscanf(fp, "%d", &temp1) == 1) {
        (*input1)[counter] = temp1;

        counter++;
    }

    fclose(fp);
    return len;

}

int read_input_three(int** input1, int** input2, int** input3, int** input4,char* filepath){
    FILE* fp = fopen(filepath, "r");
    if (fp == NULL){
     fprintf(stderr, "Couldn't open file for reading\n");
     exit(1);
    }

    int counter = 0;
    int len;
    int length = fscanf(fp, "%d", &len);
    *input1 = ( int *)malloc(len * sizeof(int));
    *input2 = ( int *)malloc(len * sizeof(int));
    *input3 = ( int *)malloc(len * sizeof(int));
    *input4 = ( int *)malloc(len * sizeof(int));

    int temp1;
    int temp2;
    int temp3;
    int temp4;
    while (fscanf(fp, "%d,%d,%d,%d", &temp1, &temp2, &temp3, &temp4) == 4) {
        (*input1)[counter] = temp1;
        (*input2)[counter] = temp2;
        (*input3)[counter] = temp3;
        (*input4)[counter] = temp4;
        counter++;
    }

    fclose(fp);
    return len;

}

__device__ int gate_solver(int gate, int A, int B) {
    switch (gate) {
        case 0: return A && B;        // AND
        case 1: return A || B;        // OR
        case 2: return !(A && B);     // NAND
        case 3: return !(A || B);     // NOR
        case 4: return A ^ B;         // XOR
        case 5: return !(A ^ B);      // XNOR
        default: return -1;           // Invalid gate type
    }
}

// Global counter for the next level nodes
__device__ int numNextLevelNodes_d = 0; 

// Helper function to add a node to the global queue
__device__ void addToGlobalQueue(int node, int *nextLevelNodes_d) {
    int index = atomicAdd(&numNextLevelNodes_d, 1);
    nextLevelNodes_d[index] = node;
}

__global__ void blockQueuingKernel(int elementsPerThread, int blockNum, int threadNum, int blockQueueCapacity,
                  int *nodePtrs_d, int *nodeGate_d, int numCurrLevelNodes, int *numNextLevelNodes_h,
                  int *currLevelNodes_d, int *nodeNeighbors_d, int *nodeVisited_d,
                  int *nodeInput_d, int *nodeOutput_d, int *nextLevelNodes_d)
{

    // Initialize shared memory for the queue and its size
    extern __shared__ int sharedQueue[];
    __shared__ int sharedQueueSize;

    // Determine the thread ID within its block, the block ID within the grid 
    // and calculate the global index for each thread
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int i = tid + (bid * blockDim.x);

    // Initialize the shared queue size to 0 by the first thread of the block
    if (tid == 0) sharedQueueSize = 0;
    __syncthreads();

    // Each thread processes multiple elements based on 'elementsPerThread' (Loop over all nodes in the current level)
    for (int idx = i * elementsPerThread; idx < (i+1) * elementsPerThread && idx < numCurrLevelNodes; idx++) {
        
        // Get the current node from the list of current level nodes
        int currNode = currLevelNodes_d[idx];
        
        // Loop over all neighbors of the node
        for (int nbrIdx = nodePtrs_d[currNode]; nbrIdx < nodePtrs_d[currNode + 1]; nbrIdx++) {
            int neighbor = nodeNeighbors_d[nbrIdx];

            // If the neighbor hasn't been visited yet
            if (!nodeVisited_d[neighbor]) {

                // Mark the neighbor as visited
                nodeVisited_d[neighbor] = 1;

                // Compute the output value for the neighbor node
                nodeOutput_d[neighbor] = gate_solver(nodeGate_d[neighbor], nodeOutput_d[currNode], nodeInput_d[neighbor]);

                // Atomically add the neighbor to the shared queue, checking for overflow
                int queueIndex = atomicAdd(&sharedQueueSize, 1);

                // If the queue is not full, add the neighbor to the shared queue
                if (queueIndex < blockQueueCapacity) {
                    sharedQueue[queueIndex] = neighbor;

                // If the queue is full, undo the last increment and add to the global queue
                } else {
                    atomicSub(&sharedQueueSize, 1);
                    addToGlobalQueue(neighbor, nextLevelNodes_d);
                }
            }
        }

        // Synchronize threads to ensure all have processed their nodes
        __syncthreads();

        // Each thread adds its portion of the shared queue to the global queue
        if (tid < sharedQueueSize) {
            addToGlobalQueue(sharedQueue[tid], nextLevelNodes_d);
        }

        // Synchronize threads to ensure global queue update is complete
        __syncthreads();
    }
}

int main(int argc, char *argv[]) {

  // Variables
  int numNodePtrs;
  int numNodes;
  int *nodePtrs_h;
  int *nodeNeighbors_h;
  int *nodeVisited_h;
  int numTotalNeighbors;
  int *currLevelNodes_h;
  int numCurrLevelNodes;
  int *numNextLevelNodes_h = 0; // Initialize to 0
  int *nodeGate_h;
  int *nodeInput_h;
  int *nodeOutput_h;
  int *nextLevelNodes_h; // Output

  // Parse command line arguments for block queuing
  int blockSize = atoi(argv[1]);
  int numBlocks = atoi(argv[2]);
  int sharedQueueSize = atoi(argv[3]);
  char* input1_path = argv[4];
  char* input2_path = argv[5];
  char* input3_path = argv[6];
  char* input4_path = argv[7];
  char* output_node_path = argv[8];
  char* output_nextnodes_path = argv[9];
  FILE* output_node;
  FILE* output_nextnodes;

  numNodePtrs = read_input_one_two_four(&nodePtrs_h, input1_path);
  numTotalNeighbors = read_input_one_two_four(&nodeNeighbors_h, input2_path);
  numNodes = read_input_three(&nodeVisited_h, &nodeGate_h, &nodeInput_h, &nodeOutput_h, input3_path);
  numCurrLevelNodes = read_input_one_two_four(&currLevelNodes_h, input4_path);

  nextLevelNodes_h = (int*)malloc(numNodePtrs * sizeof(int));

  // Init Cuda variables
  int *nodePtrs_d;
  int *nodeNeighbors_d;
  int *nodeVisited_d;
  int *currLevelNodes_d;
  int *nodeGate_d;
  int *nodeInput_d;
  int *nodeOutput_d;
  int *nextLevelNodes_d;

  hipMalloc(&currLevelNodes_d, numCurrLevelNodes * sizeof(int));
  hipMalloc(&nodePtrs_d, numNodePtrs * sizeof(int));
  hipMalloc(&nodeNeighbors_d, numTotalNeighbors * sizeof(int));
  hipMalloc(&nodeVisited_d, numNodes * sizeof(int));
  hipMalloc(&nodeGate_d, numNodes * sizeof(int));
  hipMalloc(&nodeInput_d, numNodes * sizeof(int));
  hipMalloc(&nodeOutput_d, numNodes * sizeof(int));
  hipMalloc(&nextLevelNodes_d, numNodes * sizeof(int));

  hipMemcpy(currLevelNodes_d, currLevelNodes_h, numCurrLevelNodes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nodePtrs_d, nodePtrs_h, numNodePtrs * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nodeNeighbors_d, nodeNeighbors_h, numTotalNeighbors * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nodeVisited_d, nodeVisited_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nodeGate_d, nodeGate_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nodeInput_d, nodeInput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nodeOutput_d, nodeOutput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nextLevelNodes_d, nextLevelNodes_h, numNodes * sizeof(int), hipMemcpyHostToDevice);

	hipMallocManaged(&numNextLevelNodes_h, sizeof(int));

  int sharedMemSize = sharedQueueSize * sizeof(int);

  int elementsPerThread = (numCurrLevelNodes + (numBlocks * blockSize) - 1) / (numBlocks * blockSize);

  GpuTimer timer;
  timer.Start();

  blockQueuingKernel <<<numBlocks, blockSize, sharedMemSize>>> (
    elementsPerThread, numBlocks, blockSize, sharedQueueSize, 
    nodePtrs_d, nodeGate_d, numCurrLevelNodes, numNextLevelNodes_h, 
    currLevelNodes_d, nodeNeighbors_d, nodeVisited_d, nodeInput_d, 
    nodeOutput_d, nextLevelNodes_d);
    
  hipDeviceSynchronize();

  timer.Stop();
  printf("Time Elapsed: %g ms\n", timer.Elapsed());

	//free cuda memory
	hipMemcpy(nodeOutput_h, nodeOutput_d, (numNodePtrs-1) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(nextLevelNodes_h, nextLevelNodes_d, *numNextLevelNodes_h * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(currLevelNodes_d);
	hipFree(nodePtrs_d);
	hipFree(nodeNeighbors_d);
	hipFree(nodeVisited_d);
	hipFree(nodeGate_d);
	hipFree(nodeInput_d);
	hipFree(nodeOutput_d);
	hipFree(nextLevelNodes_d);

  output_node = fopen(output_node_path, "w");
  if (output_node == NULL) {
      printf("Can't open %s", output_node_path);
      exit(1);
  }

  output_nextnodes = fopen(output_nextnodes_path, "w");
  if (output_nextnodes == NULL) {
      printf("Can't open %s", output_nextnodes_path);
      exit(1);
  }

  fprintf(output_node, "%d\n", numNodePtrs - 1);
  for (int i = 0; i < numNodePtrs - 1; i++) {
      fprintf(output_node, "%d\n", nodeOutput_h[i]);
  }

  fclose(output_node);

  fprintf(output_nextnodes, "%d\n", *numNextLevelNodes_h);
  for (int j = 0; j < *numNextLevelNodes_h; j++) {
      fprintf(output_nextnodes, "%d\n", nextLevelNodes_h[j]);
  }

  fclose(output_nextnodes);
  free(nextLevelNodes_h);

  return 0;
}
